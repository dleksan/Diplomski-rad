#include "hip/hip_runtime.h"
﻿
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include "hip/hip_runtime.h"
#include ""
#include <chrono> 
#include <iostream>
using namespace std;

#define BLOCK_SIZE 16
const int matrixSize = 512;




//Mnozenje matrica
__global__ void gpu_matrix_mult(int* A, int* B, int* C, int matrixSize)
{

    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int sum = 0;

    if (col < matrixSize && row < matrixSize)
    {
        for (int i = 0; i < matrixSize; i++)
        {
            
            sum += A[row * matrixSize + i] * B[i * matrixSize + col];
        }

        
        C[row * matrixSize + col] = sum;
    }
}



void cpu_matrix_mult(int* A, int* B, int* C, int matrixSize) {
    for (int i = 0; i < matrixSize; ++i)
    {
        for (int j = 0; j < matrixSize; ++j)
        {
            int tmp = 0.0;
            for (int h = 0; h < matrixSize; ++h)
            {
                tmp += A[i * matrixSize + h] * B[h * matrixSize + j];
            }
            C[i * matrixSize + j] = tmp;
        }
    }
}



//Popunjavanje matrica
void initializeMatrices(int matrixSize, int* A, int* B)
{
    for (int i = 0; i < matrixSize; ++i) {
        for (int j = 0; j < matrixSize; ++j) {
            A[i * matrixSize + j] = rand() % 1024;
        }
    }


    for (int i = 0; i < matrixSize; ++i) {
        for (int j = 0; j < matrixSize; ++j) {
            B[i * matrixSize + j] = rand() % 1024;
        }
    }

}


//Provjera tocnosti
bool isValid(int matrixSize, int* hC, int* dC)
{

    for (int i = 0; i < matrixSize; ++i)
    {
        for (int j = 0; j < matrixSize; ++j)
        {
            
            if (dC[i * matrixSize + j] != hC[i * matrixSize + j])
            {
                return false;
            }
        }
        
    }

    return true;
}



int main(int argc, char const* argv[])
{



//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
//                                          GPU verzija                                                                    //
//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

    int* hA, * hB, * hC, * hCC;
    
    hipHostMalloc((void**)&hA, sizeof(int) * matrixSize * matrixSize);
    hipHostMalloc((void**)&hB, sizeof(int) * matrixSize * matrixSize);
    hipHostMalloc((void**)&hC, sizeof(int) * matrixSize * matrixSize);

    //Za CPU rezultat
    hipHostMalloc((void**)&hCC, sizeof(int) * matrixSize * matrixSize);

   
    initializeMatrices(matrixSize, hA, hB);
 

    float GPU_time;
   

    // Alociranje memorije na GPU-u
    int* dA, * dB, * dC;
    hipMalloc((void**)&dA, sizeof(int) * matrixSize * matrixSize);
    hipMalloc((void**)&dB, sizeof(int) * matrixSize * matrixSize);
    hipMalloc((void**)&dC, sizeof(int) * matrixSize * matrixSize);


    //Pokretanje timera
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start, 0);


    //Kopiranje podataka na device
    hipMemcpy(dA, hA, sizeof(int) * matrixSize * matrixSize, hipMemcpyHostToDevice);
    hipMemcpy(dB, hB, sizeof(int) * matrixSize * matrixSize, hipMemcpyHostToDevice);

    unsigned int gridSize = (matrixSize + BLOCK_SIZE - 1) / BLOCK_SIZE;
    
    dim3 dimGrid(gridSize, gridSize);
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
  
    //Poziv CUDA funkcije
    gpu_matrix_mult << <dimGrid, dimBlock >> > (dA, dB, dC, matrixSize);


    //Kopiranje podataka na host-a
    hipMemcpy(hC, dC, sizeof(int) * matrixSize * matrixSize, hipMemcpyDeviceToHost);

    hipDeviceSynchronize();
   
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);  

    hipEventElapsedTime(&GPU_time, start, stop);
    
    cout << "Vrijeme na GPU:" << GPU_time << "ms" << endl;



    //////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    //                                          CPU verzija                                                                    //
    //////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

    auto clock_start_CPU = std::chrono::system_clock::now();

    cpu_matrix_mult(hA, hB, hCC, matrixSize);


    auto clock_now_CPU = std::chrono::system_clock::now();

    float CPU_time = float(std::chrono::duration_cast 
    <std::chrono::milliseconds> (clock_now_CPU - clock_start_CPU).count());


    std::cout << "Vrijeme na CPU: " << CPU_time << " ms \n";


   //////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    //                                          Provjera tocnosti                                                                  //
    /////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    if (isValid(matrixSize, hC, hCC))
    {
        
        cout << "Rezultat je tocan!" << endl;
    }
    else
    {
        cout << "Rezultati nije tocan" << endl;
        
    }

    // oslobodi memoriju
    hipFree(dA);
    hipFree(dB);
    hipFree(dC);
    hipHostFree(hA);
    hipHostFree(hB);
    hipHostFree(hC);
    hipHostFree(hCC);
    return 0;
}