#include "hip/hip_runtime.h"
﻿#include <stdio.h>
#include <iostream>
#include <fstream>
#include <vector>
#include <string>

#pragma comment(lib, "cuda.lib")
#pragma comment(lib, "cudart.lib")
#include <hip/hip_runtime.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include ""
#include <hipblas.h>

using namespace std;

#define BLOCK_SIZE 16

const int n = 512;



//Popunjavanje matrice
void initializeMatrix(double* L, int matrixSize) {
	int row, col;
	
	
	srand(3333);
	for (row = 0; row < matrixSize; row++) {
		for (col = 0; col < matrixSize; col++)
		{
			L[row * matrixSize + col]= rand()%10;
		
		}
	}
	
}

void initializeIdentityMatrix(double* I, int matrixSize)
{

	int row, col;
	for (row = 0; row < matrixSize; row++) {
		for (col  = 0; col < matrixSize; col++) {
			if (row == col) I[row * matrixSize + row] = 1.0;
			else I[row *matrixSize + col] = 0.0;
		}
	}
}




//Funkcije za računanje inverza
__global__ void nodiag_normalize(double* A, double* I, int n, int i) {
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;

	if (x < n && y < n)
		if (x == i && x != y) {
			I[x * n + y] /= A[i * n + i];
			A[x * n + y] /= A[i * n + i];
		}

}




__global__ void diag_normalize(double* A, double* I, int n, int i) {
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;

	if (x < n && y < n)
		if (x == y && x == i) {
			I[x * n + y] /= A[i * n + i];
			A[x * n + y] /= A[i * n + i];
		}

}

__global__ void gaussjordan(double* A, double* I, int n, int i)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;

	if (x < n && y < n) {
		if (x != i) {
			I[x * n + y] -= I[i * n + y] * A[x * n + i];
			if (y != i) {
				A[x * n + y] -= A[i * n + y] * A[x * n + i];
			}
		}
	}

}

__global__ void set_zero(double* A, double* I, int n, int i) {
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;

	if (x < n && y < n) {
		if (x != i) {
			if (y == i) {
				A[x * n + y] = 0;
			}
		}
	}
}



//Funkcije za ispis
void printMatrix(double* L, int matrixSize) {

	int row, col;
	for (row = 0; row < matrixSize; row++) {
		for (col = 0; col < matrixSize; col++)
		{
			cout << L[row * matrixSize + col] << " ";
		}
			
		cout << endl;
		
	}
}

void printInverse(double* iL, int dimension)
{
	int row, col;
	for (row = 0; row < dimension; row++) {
		for (col = 0; col < dimension; col++)
		{
			cout << iL[row * dimension + col] << " ";
		}

		cout << endl;

	}
}



bool isValid(double *L, double *iL)
{
	int row, col;
	bool flag = true;
	double* c = new double[n * n];

	for (row = 0; row < n; row++)
	{
		for (col = 0; col < n; col++)
		{
			c[row * n + col] = 0;  
			for (int x = 0; x < n; x++)
			{
				c[row * n + col] += L[row * n + x] * iL[x * n + col]; 

			}

			
		}

	}

	//Provjerja je li matrica s lijeve strane jedinična
	for (row = 0; row < n; row++)
	{
		for (col = 0; col < n; col++)
		{
			if (row == col)
			{
				if (c[row * n + col] >1+ 1e-09 || c[row * n + col] < 1-1e-09)
				{
					
					
					flag = false;
					
				}
				//cout << c[row * n + col]<< " ";
				//printf("%.16f", c[row * n + col]);
			}

			else
			{
				if (c[row * n + col] > 1e-09 || c[row * n + col] < -1e-09)
				{
					
					flag = false;
					
				}
			}

		}



	}

	return flag;

}


int main()
{
	
	// creating input
	double* iL = new double[n * n];
	double* L = new double[n * n];
	initializeMatrix(&L[0], n);
	
	//printMatrix(&L[0], n);

	
	double* d_A, * d_L, * I, * d_I;
	float time;
	hipError_t err;
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	int allocationSize = n * n * sizeof(double);

	dim3 threadsPerBlock(BLOCK_SIZE, BLOCK_SIZE);
	dim3 numBlocks((n + BLOCK_SIZE - 1) / BLOCK_SIZE, (n + BLOCK_SIZE - 1) / BLOCK_SIZE);

	// alokacija memorije    
	hipMalloc((void**)&d_A, allocationSize);
	
	hipMalloc((void**)&d_I, allocationSize);
	
	I = new double[n * n];


	initializeIdentityMatrix(I, n);
	
	//Pokretanje timera
	hipEventRecord(start, 0);

	//kopiranje s hosta na device
	hipMemcpy(d_A, L, allocationSize, hipMemcpyHostToDevice);
	
    hipMemcpy(d_I, I, allocationSize, hipMemcpyHostToDevice);
	


	// Racunanje inverza    
	for (int i = 0; i < n; i++) {
		nodiag_normalize << <numBlocks, threadsPerBlock >> > (d_A, d_I, n, i);
		diag_normalize << <numBlocks, threadsPerBlock >> > (d_A, d_I, n, i);
		gaussjordan << <numBlocks, threadsPerBlock >> > (d_A, d_I, n, i);
		set_zero << <numBlocks, threadsPerBlock >> > (d_A, d_I, n, i);
	}


	//kopiranje s devicea na hosta
	hipMemcpy(iL, d_I, allocationSize, hipMemcpyDeviceToHost);
	
	hipMemcpy(I, d_A, allocationSize, hipMemcpyDeviceToHost);
	

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&time, start, stop);
	hipEventDestroy(start);
	hipEventDestroy(stop);

	cout << "Vrijeme: " << time << "ms\n";
	
	
	//printMatrix(L, n);
	//printInverse(iL,n);

	/////////////////////////////////////////////////////////////////
	///						Provjera tocnosti					  ///
	/////////////////////////////////////////////////////////////////
	if (isValid(L, iL))
	{
		cout << "Rezultat je tocan!" << endl;
	}

	else {
		cout << "Rezultat je netocan" << endl;
	}
	



	hipFree(d_A);
	hipFree(d_I);

	
	

	delete[]I;
	delete[]L;
	delete[]iL;

	system("Pause");
	return 0;
}