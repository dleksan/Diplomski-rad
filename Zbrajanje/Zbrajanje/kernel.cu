#include "hip/hip_runtime.h"
﻿#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include ""
#include <chrono>
#include <ctime>
using namespace std;


#define matrixSize 3000


//int** A=NULL;
//int** B=NULL;
//int** C_GPU= NULL;
//int** C_CPU= NULL;
#define BLOCK_SIZE 50


//void allocateMatrices()
//{
//    A = new int* [matrixSize];
//    B = new int* [matrixSize];
//    C_GPU = new int* [matrixSize];
//    C_CPU = new int* [matrixSize];
//
//    for (int i = 0; i < matrixSize; i++) {
//
//        // Declare a memory block
//        // of size n
//        A[i] = new int[matrixSize];
//        B[i] = new int[matrixSize];
//        C_GPU[i] = new int[matrixSize];
//        C_CPU[i] = new int[matrixSize];
//    }
//
//}




int A[matrixSize][matrixSize];
int B[matrixSize][matrixSize];
int C_GPU[matrixSize][matrixSize];
int C_CPU[matrixSize][matrixSize];




 //Matrix addition kernel
__global__ void matrixAddition_GPU(int* A, int* B, int* C) {
    
    int x = blockIdx.x;
    int y = blockIdx.y;
    int id = gridDim.x * y + x;
    C[id] = A[id] + B[id];

    /*int x = blockIdx.x + threadIdx.x;
    int y= blockIdx.y + threadIdx.y;
    int id =gridDim.x * y + x;
    C[id] = A[id] + B[id];
    */
    
}

void matrixAddition_CPU() {

    int row, col;
    for (row = 0; row < matrixSize; row++)
    {
        for (col = 0; col < matrixSize; col++)
        {
            C_CPU[row][col] = A[row][col] + B[row][col];
        }
    }
}



void fillMatrices()
{
    int row, col;
    for (row = 0; row < matrixSize; row++)
    {
        for (col = 0; col < matrixSize; col++)
        {
            A[row][col] = rand() % 10;
            B[row][col] = rand() % 10;
        }
    }
}

void printResult()
{
    cout << "Result matrix:" << endl;
    int row, col;
    for (row = 0; row < matrixSize; row++)
    {
        for (col = 0; col < matrixSize; col++)
        {
            cout<<C_GPU[row][col]<<" ";
        }

        cout << endl;
    }


    cout << "Na CPU:" << endl;
    for (row = 0; row < matrixSize; row++)
    {
        for (col = 0; col < matrixSize; col++)
        {
            cout << C_CPU[row][col] << " ";
        }

        cout << endl;
    }
}

void printMatrices()
{

    int row, col;

    cout << "First matrix:" << endl;
    for (row = 0; row < matrixSize; row++)
    {
        for (col = 0; col < matrixSize; col++)
        {
            cout << A[row][col]<<" ";
        }

        cout << endl;
    }


    cout << "Second matrix:" << endl;
    for (row = 0; row < matrixSize; row++)
    {
        for (col = 0; col < matrixSize; col++)
        {
            cout << B[row][col] << " ";
        }

        cout << endl;
    }
}

bool isValid()
{
    int row, col;
    for (row = 0; row < matrixSize; row++)
    {
        for (col = 0; col < matrixSize; col++)
        {
            if (C_GPU[row][col] != C_CPU[row][col]) {
                return false;
            }
        }
    }

    return true;
}

int main() {
   
    ////////////////////////////////////////////
    //               GPU                     //
    ////////////////////////////////////////////
    float time_GPU;
    int* A_d, * B_d, * C_d;
    
	hipEvent_t start_GPU, stop_GPU;
	hipEventCreate(&start_GPU);
	hipEventCreate(&stop_GPU);

    //allocateMatrices();
    fillMatrices();

    //printMatrices();

   

    hipMalloc((void**)&A_d, matrixSize * matrixSize * sizeof(int));
    hipMalloc((void**)&B_d, matrixSize * matrixSize * sizeof(int));
    hipMalloc((void**)&C_d, matrixSize * matrixSize * sizeof(int));

    hipMemcpy(A_d, A, matrixSize * matrixSize * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(B_d, B, matrixSize * matrixSize * sizeof(int), hipMemcpyHostToDevice);

   
   /* dim3 dimBlock(BLOCK_SIZE,BLOCK_SIZE);
    
    dim3 dimGrid((matrixSize + BLOCK_SIZE - 1) / BLOCK_SIZE, (matrixSize + BLOCK_SIZE - 1) / BLOCK_SIZE);*/

    dim3 dimGrid(matrixSize, matrixSize);
    hipEventRecord(start_GPU, 0);

                            //grid matrix size i 1 thread
    matrixAddition_GPU << < dimGrid, 1 >> > (A_d, B_d, C_d);

    hipEventRecord(stop_GPU, 0);
    hipMemcpy(C_GPU, C_d, matrixSize * matrixSize * sizeof(int), hipMemcpyDeviceToHost);
    
	hipEventSynchronize(stop_GPU);
	hipEventElapsedTime(&time_GPU, start_GPU, stop_GPU);
	hipEventDestroy(start_GPU);
	hipEventDestroy(stop_GPU);

    cout << "Vrijeme na GPU: " << time_GPU << "ms\n";

    

    //////////////////////////////////////////////////
      //                    CPU                       //
    //////////////////////////////////////////////////

    

    auto start_CPU = chrono::high_resolution_clock::now();
    matrixAddition_CPU();
    auto stop_CPU = chrono::high_resolution_clock::now();

    auto time_CPU = chrono::duration_cast<chrono::milliseconds>(stop_CPU - start_CPU);

    cout << "Vrijeme na CPU:" << time_CPU.count() << endl;;

    ///////////////////////////////////////////////
     //             Provjera ispravnosti       //
   ////////////////////////////////////////////////

    //printMatrices();
   // printResult();

    if (isValid())
    {
        cout << "Rezultat je tocan!";
    }

    else
    {
        cout << "Rezultat je netocan!";
    }


    hipFree(A_d);
    hipFree(B_d);
    hipFree(C_d);
    return 0;

    
}

//#include <iostream>
//#include <hip/hip_runtime.h>
//#include <hip/hip_runtime_api.h>
//#include ""
//#include <chrono>
//
//using namespace std;
//
//#define matrixSize 100
//
//int A[matrixSize][matrixSize];
//int B[matrixSize][matrixSize];
//int C_GPU[matrixSize][matrixSize];
//int C_CPU[matrixSize][matrixSize];
//
//using namespace std;
//
//__global__ void matrixAddition_GPU(int A[][matrixSize], int B[][matrixSize], int C_GPU[][matrixSize]) {
//
//    int i = threadIdx.x;
//    int j = threadIdx.y;
//
//    C_GPU[i][j] = A[i][j] + B[i][j];
//}
//
//
//void matrixAddition_CPU() {
//
//    int row, col;
//    for (row = 0; row < matrixSize; row++)
//    {
//        for (col = 0; col < matrixSize; col++)
//        {
//            C_CPU[row][col] = A[row][col] + B[row][col];
//        }
//    }
//}
//
//void fillMatrices()
//{
//    int row, col;
//    for (row = 0; row < matrixSize; row++)
//    {
//        for (col = 0; col < matrixSize; col++)
//        {
//            A[row][col] = rand() % 10;
//            B[row][col] = rand() % 10;
//        }
//    }
//}
//
//bool isValid()
//{
//    int row, col;
//    for (row = 0; row < matrixSize; row++)
//    {
//        for (col = 0; col < matrixSize; col++)
//        {
//            if (C_GPU[row][col] != C_CPU[row][col]) {
//                return false;
//            }
//        }
//    }
//
//    return true;
//}
//
//void printResult()
//{
//    cout << "Result matrix:" << endl;
//    int row, col;
//    cout << "GPU:"<<endl;
//    for (row = 0; row < matrixSize; row++)
//    {
//        for (col = 0; col < matrixSize; col++)
//        {
//            cout<<C_GPU[row][col]<<" ";
//        }
//
//        cout << endl;
//    }
//
//    cout << "CPU:" << endl;
//    for (row = 0; row < matrixSize; row++)
//    {
//        for (col = 0; col < matrixSize; col++)
//        {
//            cout << C_CPU[row][col] << " ";
//        }
//
//        cout << endl;
//    }
//}
//
//int main() {
//
//    //////////////////////////////////////////////////////////
//    //////////              GPU                     //////////
//    //////////////////////////////////////////////////////////
// 
//    hipEvent_t start_GPU, stop_GPU;
//	hipEventCreate(&start_GPU);
//	hipEventCreate(&stop_GPU);
//
//    float time_GPU;
//
//
//    int(*A_d)[matrixSize], (*B_d)[matrixSize], (*C_d)[matrixSize];
//
//    fillMatrices();
//
//    hipMalloc((void**)&A_d, (matrixSize * matrixSize) * sizeof(int));
//    hipMalloc((void**)&B_d, (matrixSize * matrixSize) * sizeof(int));
//    hipMalloc((void**)&C_d, (matrixSize * matrixSize) * sizeof(int));
//
//
//    hipEventRecord(start_GPU, 0);
//
//    hipMemcpy(A_d, A, (matrixSize * matrixSize) * sizeof(int), hipMemcpyHostToDevice);
//    hipMemcpy(B_d, B, (matrixSize * matrixSize) * sizeof(int), hipMemcpyHostToDevice);
//    hipMemcpy(C_d, C_GPU, (matrixSize * matrixSize) * sizeof(int), hipMemcpyHostToDevice);
//
//    int numBlocks = 1;
//    dim3 threadsPerBlock(matrixSize, matrixSize);
//    matrixAddition_GPU << < 1, threadsPerBlock>> > (A_d, B_d, C_d);
//
//    hipMemcpy(C_GPU, C_d, (matrixSize * matrixSize) * sizeof(int), hipMemcpyDeviceToHost);
//    hipEventRecord(stop_GPU, 0);
//   
//    hipEventSynchronize(stop_GPU);
//	hipEventElapsedTime(&time_GPU, start_GPU, stop_GPU);
//	hipEventDestroy(start_GPU);
//	hipEventDestroy(stop_GPU);
//
//    cout << "Vrijeme na GPU: " << time_GPU << "ms\n";
//
//        //////////////////////////////////////////////////
//       //                   CPU                       //
//    //////////////////////////////////////////////////
//
//    
//
//    auto start_CPU = chrono::high_resolution_clock::now();
//    matrixAddition_CPU();
//    auto stop_CPU = chrono::high_resolution_clock::now();
//
//    auto time_CPU = chrono::duration_cast<chrono::microseconds>(stop_CPU - start_CPU);
//
//    cout << "Vrijeme na CPU:" << time_CPU.count() << endl;;
//
//    ///////////////////////////////////////////////
//       //               Provjera ispravnosti       //
//   ////////////////////////////////////////////////
//
//    if (isValid())
//    {
//        cout << "Rezultat je tocan!";
//    }
//
//    else
//    {
//        cout << "Rezultat je netocan!";
//    }
//
//    printResult();
//
//    hipFree(A_d);
//    hipFree(B_d);
//    hipFree(C_d);
//
//    printf("\n");
//
//    return 0;
//}