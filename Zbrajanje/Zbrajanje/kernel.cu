#include "hip/hip_runtime.h"
﻿
#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include ""
#include <chrono>
#include <ctime>
using namespace std;


#define matrixSize 3000



int A[matrixSize][matrixSize];
int B[matrixSize][matrixSize];
int C_GPU[matrixSize][matrixSize];
int C_CPU[matrixSize][matrixSize];




//Zbrajanje matrica
__global__ void matrixAddition_GPU(int* A, int* B, int* C) {

    int x = blockIdx.x;
    int y = blockIdx.y;
    int id = gridDim.x * y + x;
    C[id] = A[id] + B[id];

}


void matrixAddition_CPU() {

    int row, col;
    for (row = 0; row < matrixSize; row++)
    {
        for (col = 0; col < matrixSize; col++)
        {
            C_CPU[row][col] = A[row][col] + B[row][col];
        }
    }
}


//Popunjavanje matrica
void fillMatrices()
{
    int row, col;
    for (row = 0; row < matrixSize; row++)
    {
        for (col = 0; col < matrixSize; col++)
        {
            A[row][col] = rand() % 10;
            B[row][col] = rand() % 10;
        }
    }
}


//Ispis matrica
void printResult()
{
    cout << "Result matrix:" << endl;
    int row, col;
    for (row = 0; row < matrixSize; row++)
    {
        for (col = 0; col < matrixSize; col++)
        {
            cout << C_GPU[row][col] << " ";
        }

        cout << endl;
    }


    cout << "Na CPU:" << endl;
    for (row = 0; row < matrixSize; row++)
    {
        for (col = 0; col < matrixSize; col++)
        {
            cout << C_CPU[row][col] << " ";
        }

        cout << endl;
    }
}



void printMatrices()
{

    int row, col;

    cout << "First matrix:" << endl;
    for (row = 0; row < matrixSize; row++)
    {
        for (col = 0; col < matrixSize; col++)
        {
            cout << A[row][col] << " ";
        }

        cout << endl;
    }


    cout << "Second matrix:" << endl;
    for (row = 0; row < matrixSize; row++)
    {
        for (col = 0; col < matrixSize; col++)
        {
            cout << B[row][col] << " ";
        }

        cout << endl;
    }
}


//Provjera tocnosti
bool isValid()
{
    int row, col;
    for (row = 0; row < matrixSize; row++)
    {
        for (col = 0; col < matrixSize; col++)
        {
            if (C_GPU[row][col] != C_CPU[row][col]) {
                return false;
            }
        }
    }

    return true;
}

int main() {

    ////////////////////////////////////////////
    //               GPU verzija             //
    ////////////////////////////////////////////
    float time_GPU;
    int* A_d, * B_d, * C_d;

    hipEvent_t start_GPU, stop_GPU;
    hipEventCreate(&start_GPU);
    hipEventCreate(&stop_GPU);

    
    fillMatrices();

    //printMatrices();


    //Alociranje memorije na GPU-u
    hipMalloc((void**)&A_d, matrixSize * matrixSize * sizeof(int));
    hipMalloc((void**)&B_d, matrixSize * matrixSize * sizeof(int));
    hipMalloc((void**)&C_d, matrixSize * matrixSize * sizeof(int));


    //Pokretanje timera
    hipEventRecord(start_GPU, 0);

    hipMemcpy(A_d, A, matrixSize * matrixSize * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(B_d, B, matrixSize * matrixSize * sizeof(int), hipMemcpyHostToDevice);



    dim3 dimGrid(matrixSize, matrixSize);


    //Pozivanje CUDA funkcije
    matrixAddition_GPU << < dimGrid, 1 >> > (A_d, B_d, C_d);


    hipMemcpy(C_GPU, C_d, matrixSize * matrixSize * sizeof(int), hipMemcpyDeviceToHost);
    hipEventRecord(stop_GPU, 0);
    hipEventSynchronize(stop_GPU);
    hipEventElapsedTime(&time_GPU, start_GPU, stop_GPU);
    hipEventDestroy(start_GPU);
    hipEventDestroy(stop_GPU);

    cout << "Vrijeme na GPU: " << time_GPU << "ms\n";



    //////////////////////////////////////////////////
    //                    CPU verzija               //
    //////////////////////////////////////////////////
    auto start_CPU = chrono::high_resolution_clock::now();
    matrixAddition_CPU();
    auto stop_CPU = chrono::high_resolution_clock::now();

    auto time_CPU = chrono::duration_cast<chrono::milliseconds>(stop_CPU - start_CPU);

    cout << "Vrijeme na CPU:" << time_CPU.count() << endl;;


    ///////////////////////////////////////////////
    //             Provjera točnosti            //
   ////////////////////////////////////////////////

    //printMatrices();
   // printResult();

    if (isValid())
    {
        cout << "Rezultat je tocan!";
    }

    else
    {
        cout << "Rezultat je netocan!";
    }


    hipFree(A_d);
    hipFree(B_d);
    hipFree(C_d);
    return 0;


}